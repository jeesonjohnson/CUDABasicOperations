#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16
// Matrices are stored in row-major order
typedef struct {
 int width;
 int height;
 float* elements;
} Matrix;

__global__ void MatrixMultKern(const Matrix A, const Matrix B, const Matrix C) {
    // Calculate the column index of C and B
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the row index of C and of A
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < A.height) && (col < B.width)) {
    float Cvalue = 0;
    // each thread computes one element of the block sub-matrix
    for (int k = 0; k < A.width; ++k) {
    Cvalue += A.elements[row * A.width + k] * B.elements[k*B.width + col];
    }
    C.elements[row * C.width + col] = Cvalue;
    }
   }


   // Matrix multiplication - Host Code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatrixMult(const Matrix h_A, const Matrix h_B, Matrix h_C)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


 // Load A and B into device memory
 Matrix d_A;
 d_A.width = h_A.width; d_A.height = h_A.height;
 size_t size = h_A.width * h_A.height * sizeof(float);
 hipMalloc(&d_A.elements, size);
 hipMemcpy(d_A.elements, h_A.elements, size, hipMemcpyHostToDevice);
 Matrix d_B;
 d_B.width = h_B.width; d_B.height = h_B.height;
 size = h_B.width * h_B.height * sizeof(float);
 hipMalloc(&d_B.elements, size);
 hipMemcpy(d_B.elements, h_B.elements, size, hipMemcpyHostToDevice);
 // Allocate C in Device memory
 Matrix d_C;
 d_C.width = h_C.width; d_C.height = h_C.height;
 size = h_C.width * h_C.height * sizeof(float);
 hipMalloc(&d_C.elements, size);
 // Invoke Kernel
 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
 dim3 dimGrid(d_B.width / dimBlock.x, d_A.height / dimBlock.y);
 hipEventRecord(start);
 MatrixMultKern<<< dimGrid, dimBlock >>>(d_A, d_B, d_C);
 hipDeviceSynchronize();
 // Recording the execution time below
 hipEventRecord(stop);
 hipEventSynchronize(stop);
 
 float totalExecutionTime = 0;
 hipEventElapsedTime(&totalExecutionTime, start, stop);
 printf("%f\n",totalExecutionTime);

 // Read C from Device to Host
 hipMemcpy(h_C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
 // Free Device Memory
 hipFree(d_A.elements);
 hipFree(d_B.elements);
 hipFree(d_C.elements);
}





void printSelectAmount(Matrix A,Matrix B,Matrix C,int N){
    printf("\t");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%f ", A.elements[i * A.width + j]);
        printf("\n\t");
    }
    printf("\n\t");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%f ", B.elements[i * B.width + j]);
        printf("\n\t");
    }
    printf("\n\t");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%f ", C.elements[i * C.width + j]);
        printf("\n\t");
    }
    printf("\n\t");
}


void mainMatrixFunction(int N){
    Matrix A, B, C;
    // Read Dimensions of A and B
    A.height = N;
    A.width = N;
    B.height = A.width;
    B.width = N;
    A.elements = (float *)malloc(A.width * A.height * sizeof(float));
    B.elements = (float *)malloc(B.width * B.height * sizeof(float));
    C.height = A.height;
    C.width = B.width;
    C.elements = (float *)malloc(C.width * C.height * sizeof(float));
    for (int i = 0; i < A.height; i++)
        for (int j = 0; j < A.width; j++)
            A.elements[i * A.width + j] = (float)(rand() % 3);
    for (int i = 0; i < B.height; i++)
        for (int j = 0; j < B.width; j++)
            B.elements[i * B.width + j] = (float)(rand() % 2);
    MatrixMult(A, B, C);
    printSelectAmount(A,B,C,N);
}

int main(int argc, char *argv[])
{
    for(int x=1;x<10;x*=2){
        // printf("%d\n",16*x);
        mainMatrixFunction(16);
    }
    // for(int x=1;x<1024;x*=2){
    //     // printf("%d\n",16*x);
    //     mainMatrixFunction(16*x);
    // }

    return 0;
}