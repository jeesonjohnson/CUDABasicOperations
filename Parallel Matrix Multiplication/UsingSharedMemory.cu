#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16
// Matrices are stored in row-major order
typedef struct
{
    int width;
    int height;
    float *elements;
    int stride;
} Matrix;
__global__ void MultKernShared(const Matrix A, const Matrix B, Matrix C);

// For simplicity you could just put all the content into this file instead of having a separate .h file
// Now, we are going to have to define some utilities to get and set
// elements in matrices and to get sub-matrices. Please refer to Lecture 13
// to make sure you understand why we need to use the "stride" property to
// walk down the column of a sub-matrix
// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}
// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value)
{
    A.elements[row * A.stride + col] = value;
}
// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
// Note, we are not doing any copying here - just finding the address of the
// start of the sub-matrix we are interested in
__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return Asub;
}
// Now we have all the device functions we need to define the matrix multiplication
// Kernel. This is going to be called from the host by MatMul()
__global__ void MultSharedKernel(Matrix A, Matrix B, Matrix C)
{
    // Identify the Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;
    // Now find the row and column of the element within Csub
    // that this thread is going to calculate
    int row = threadIdx.y;
    int col = threadIdx.x;
    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m)
    {
        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);
        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        // Load Asub and Bsub from global memory into shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronise to make sure the sub-matrices are completely loaded
        // before starting the computation for each phase
        __syncthreads();
        // Now multiply Asub and Bsub together to complete phase m of the
        // calculation of this threads element of Csub
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        // Synchronise again to make sure that the preceding calculation
        // has been completed by all threads in the block before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    // Once all the phases are complete we can write Csub to device (global) memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatrixMult(const Matrix h_A, const Matrix h_B, Matrix h_C) {
    //Functionaltiy for CUDA timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = h_A.width;
    d_A.height = h_A.height;
    size_t size = h_A.width * h_A.height * sizeof(float);
    hipError_t err = hipMalloc(&d_A.elements, size);
    //printf("CUDA malloc h_A: %s\n",cudaGetErrorString(err));
    hipMemcpy(d_A.elements, h_A.elements, size, hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = h_B.width;
    d_B.height = h_B.height;
    size = h_B.width * h_B.height * sizeof(float);
    err = hipMalloc(&d_B.elements, size);
    //printf("CUDA malloc h_B: %s\n",cudaGetErrorString(err));
    hipMemcpy(d_B.elements, h_B.elements, size, hipMemcpyHostToDevice);
    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = h_C.width;
    d_C.height = h_C.height;
    size = h_C.width * h_C.height * sizeof(float);
    err = hipMalloc(&d_C.elements, size);

    //printf("CUDA malloc h_C: %s\n",cudaGetErrorString(err));
    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(h_B.width / dimBlock.x, h_A.height / dimBlock.y);

    //Cuda timing functionaltiy
    hipEventRecord(start);
    MultSharedKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    //Cuda Timing functionaltiy
    hipEventRecord(stop);

    hipDeviceSynchronize();
    //printf("Run kernel: %s\n", cudaGetErrorString(err));
    // Read C from device memory
    err = hipMemcpy(h_C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    //printf("Copy h_C off device: %s\n",cudaGetErrorString(err));
    // Free device memory
    // cudaEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //printf("Elapsed time was: %f\n milliseconds", milliseconds);
    printf("%f\n",milliseconds);

    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
   }



   void printSelectAmount(Matrix A,Matrix B,Matrix C,int N){
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%f ", A.elements[i * A.width + j]);
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%f ", B.elements[i * B.width + j]);
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%f ", C.elements[i * C.width + j]);
        printf("\n");
    }
    printf("\n");
}

void mainMatrixFunction(int N){
    Matrix A, B, C;
    // Read Dimensions of A and B
    A.height = N;
    A.width = N;
    B.height = A.width;
    B.width = N;
    A.elements = (float *)malloc(A.width * A.height * sizeof(float));
    B.elements = (float *)malloc(B.width * B.height * sizeof(float));
    C.height = A.height;
    C.width = B.width;
    C.elements = (float *)malloc(C.width * C.height * sizeof(float));
    for (int i = 0; i < A.height; i++)
        for (int j = 0; j < A.width; j++)
            A.elements[i * A.width + j] = (float)(rand() % 3);
    for (int i = 0; i < B.height; i++)
        for (int j = 0; j < B.width; j++)
            B.elements[i * B.width + j] = (float)(rand() % 2);
    MatrixMult(A, B, C);
    printSelectAmount(A,B,C,N);
}

int main(int argc, char *argv[])
{
    for(int x=1;x<1024;x*=2){
        //printf("%d\n",16*x);
        mainMatrixFunction(16);
    }

    return 0;
}